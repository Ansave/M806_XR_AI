#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>

//#define N 1024
//#define N 2048
//#define N 4096
#define N 8192
//#define N 16384
//#define N 32768
//#define N 65536

using namespace std;

#define imin(a,b) (a<b?a:b)

const int threadsPerBlock = 512;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void reduction( int* in, unsigned long long int* reduced)
{
	__shared__ unsigned long long int cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	// Сложение значений входного массива
	for (int i = tid; i < N; i += gridDim.x * blockDim.x) {
		cache[threadIdx.x] += in[tid] * in[tid];
	}

	__syncthreads();

	// Непосредственно редукция
	for (int i = blockDim.x / 2; (i != 0) && (threadIdx.x < i); i /= 2) {
		cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
	}

	// Запись частичных сумм в массив меньшего размера (Результат выполнения редукции)
	if (threadIdx.x == 0) {
		reduced[blockIdx.x] = cache[0];
	}
}

int main()
{
	// Инициализация массивов
	int in[N], * dev_in;
	unsigned long long int reduced[blocksPerGrid], * dev_reduced;

	// Выделение памяти
	hipMalloc((void**)&dev_in, N * sizeof(int));
	hipMalloc((void**)&dev_reduced, blocksPerGrid * sizeof(long long int));

	// Заполнение входного массива (арифметическая прогрессия)
	for (int i = 0; i < N; i++)
	{
		in[i] = i;
	}

	// Передача данных устройству
	hipMemcpy(dev_in, in, N * sizeof(int), hipMemcpyHostToDevice);

	// Исполнение на устройстве
	reduction << < blocksPerGrid, threadsPerBlock >> > (dev_in, dev_reduced);

	// Передача данных хосту
	hipMemcpy(reduced, dev_reduced, blocksPerGrid * sizeof(long long int), hipMemcpyDeviceToHost);

	// Вывод результирующих значений
	printf("\n Count of numbers: %d", N);
	printf("\n Blocks per grid: %d", blocksPerGrid);
	printf("\n Threads per block: %d", threadsPerBlock);
	printf("\n ---");
	unsigned long long int sum = 0;
	for (int i = 0; i < blocksPerGrid; i++)
	{
		printf("\n Temp sum of %d block is %lld", i, reduced[i]);
		sum += reduced[i];
	}
	printf("\n ---");
	printf("\n Result sum = %lld\n", sum);

	// Освобождение памяти
	hipFree(dev_in);
	hipFree(dev_reduced);

	return 0;
}