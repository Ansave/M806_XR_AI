#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>

//#define N 1024
//#define N 2048
//#define N 4096
#define N 8192
//#define N 16384
//#define N 32768
//#define N 65536

using namespace std;

#define imin(a,b) (a<b?a:b)

const int threadsPerBlock = 512;
const int blocksPerGrid = imin(32, (N + threadsPerBlock - 1) / threadsPerBlock);

__global__ void calcDispertion(int* a, int* b, int* reduced)
{
	__shared__ int cache[threadsPerBlock];
	int tid = threadIdx.x + blockIdx.x * blockDim.x;

	// Вычисление суммы разностей полученных результатов от ожидаемых
	for (int i = tid; i < N; i += gridDim.x * blockDim.x) {
		cache[threadIdx.x] += (a[tid] - b[tid]) * (a[tid] - b[tid]);
	}

	__syncthreads();

	// Непосредственно редукция
	for (int i = blockDim.x / 2; (i != 0) && (threadIdx.x < i); i /= 2) {
		cache[threadIdx.x] += cache[threadIdx.x + i];
		__syncthreads();
	}

	// Запись частичных сумм в массив меньшего размера (Результат выполнения редукции)
	if (threadIdx.x == 0) {
		reduced[blockIdx.x] = cache[0];
	}
}

int main()
{
	// Инициализация массивов
	int a[N], b[N], reduced[blocksPerGrid];
	int* dev_a, * dev_b, * dev_reduced;

	// Выделение памяти
	hipMalloc((void**)&dev_a, N * sizeof(int));
	hipMalloc((void**)&dev_b, N * sizeof(int));
	hipMalloc((void**)&dev_reduced, blocksPerGrid * sizeof(int));

	// Заполнение входных массивов
	for (int i = 0; i < N; i++)
	{
		a[i] = i;
		b[i] = i + i % 2;
	}

	// Передача данных устройству
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	// Исполнение на устройстве
	calcDispertion << < blocksPerGrid, threadsPerBlock >> > (dev_a, dev_b, dev_reduced);

	// Передача данных хосту
	hipMemcpy(reduced, dev_reduced, blocksPerGrid * sizeof(int), hipMemcpyDeviceToHost);

	// Вывод результирующих значений
	printf("\n Count of numbers: %d", N);
	printf("\n Blocks per grid: %d", blocksPerGrid);
	printf("\n Threads per block: %d", threadsPerBlock);
	printf("\n ---");
	
	float disp = 0;
	for (int i = 0; i < blocksPerGrid; i++)
	{
		printf("\n Temp sum of %d block is %d", i, reduced[i]);
		disp +=  reduced[i];
	}
	disp /= N;
	
	printf("\n ---");
	printf("\n Dispertion = %f\n", disp);

	// Освобождение памяти
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_reduced);

	return 0;
}
